#include "hip/hip_runtime.h"
#include "node_noise.hpp"

#include "cuda_includes.hpp"

#include <glm/gtc/noise.hpp>

NodeNoise::NodeNoise()
    : Node("noise")
{
    addPin(PinType::OUTPUT, "image").setSingleChannel();
}

__global__ void kernNoise(Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outTex.resolution.x || y >= outTex.resolution.y)
    {
        return;
    }

    float noise = glm::simplex(glm::vec2(x, y) * 0.005f);
    outTex.setColor<TextureType::SINGLE>(x, y, noise);
}

void NodeNoise::_evaluate()
{
    Texture* outTex = nodeEvaluator->requestTexture<TextureType::SINGLE>();

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_2D_X, DEFAULT_BLOCK_SIZE_2D_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(outTex->resolution, blockSize);
    kernNoise<<<blocksPerGrid, blockSize>>>(*outTex);

    outputPins[0].propagateTexture(outTex);
}
