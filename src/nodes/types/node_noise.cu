#include "hip/hip_runtime.h"
#include "node_noise.hpp"

#include "cuda_includes.hpp"

#include <glm/gtc/noise.hpp>

NodeNoise::NodeNoise()
    : Node("noise")
{
    addPin(PinType::OUTPUT, "image");
}

__global__ void kernNoise(Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outTex.resolution.x || y >= outTex.resolution.y)
    {
        return;
    }

    float noise = glm::simplex(glm::vec2(x, y) * 0.005f);
    outTex.dev_pixels[y * outTex.resolution.x + x] = glm::vec4(glm::vec3(noise), 1);
}

void NodeNoise::evaluate()
{
    Texture* outTex = nodeEvaluator->requestTexture();

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(outTex->resolution, blockSize);
    kernNoise<<<blocksPerGrid, blockSize>>>(*outTex);

    outputPins[0].propagateTexture(outTex);
}
