#include "hip/hip_runtime.h"
#include "node_noise.hpp"

#include "cuda_includes.hpp"

#include <glm/gtc/noise.hpp>

NodeNoise::NodeNoise()
    : Node("noise")
{
    addPins(0, 1);
}

__global__ void kernNoise(Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outTex.resolution.x || y >= outTex.resolution.y)
    {
        return;
    }

    float noise = glm::simplex(glm::vec2(x, y) * 0.005f);
    outTex.dev_pixels[y * outTex.resolution.x + x] = glm::vec4(glm::vec3(noise), 1);
}

void NodeNoise::evaluate()
{
    Texture* outTex = nodeEvaluator->requestTexture();

    const dim3 blockSize(16, 16);
    const dim3 blocksPerGrid(outTex->resolution.x / 16 + 1, outTex->resolution.y / 16 + 1);
    kernNoise<<<blocksPerGrid, blockSize>>>(*outTex);

    outputPins[0].propagateTexture(outTex);

    if (getIsSelected())
    {
        nodeEvaluator->setSelectedTexture(outTex);
    }
}
