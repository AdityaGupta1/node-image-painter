#include "hip/hip_runtime.h"
#include "node_mix.hpp"

#include "cuda_includes.hpp"

NodeMix::NodeMix()
    : Node("mix")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "image 1");
    addPin(PinType::INPUT, "image 2");
    addPin(PinType::INPUT, "factor");
}

__host__ __device__ glm::vec4 mixCols(glm::vec4 col1, glm::vec4 col2, float factor)
{
    return glm::mix(col1, col2, factor);
}

__global__ void kernMix(Texture inTex1, Texture inTex2, Texture inTexFactor, glm::ivec2 outRes, Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outRes.x || y >= outRes.y)
    {
        return;
    }

    glm::vec4 col1 = inTex1.getColorClamp(x, y);
    glm::vec4 col2 = inTex2.getColorClamp(x, y);
    float factor = inTexFactor.getColorClamp(x, y).r;

    outTex.dev_pixels[y * outRes.x + x] = mixCols(col1, col2, factor);
}

bool NodeMix::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT || pin->hasEdge())
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // in color 1
        ImGui::SameLine();
        return NodeUI::ColorEdit4(constParams.color1);
    case 1: // in color 2
        ImGui::SameLine();
        return NodeUI::ColorEdit4(constParams.color2);
    case 2: // factor
        ImGui::SameLine();
        return NodeUI::FloatEdit(constParams.factor, 0.01f, 0.f, 1.f);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

// should work for differing resolutions but that hasn't been tested yet
void NodeMix::_evaluate()
{
    Texture* inTex1 = getPinTextureOrSingleColor(inputPins[0], ColorUtils::srgbToLinear(constParams.color1));
    Texture* inTex2 = getPinTextureOrSingleColor(inputPins[1], ColorUtils::srgbToLinear(constParams.color2));
    Texture* inTexFactor = getPinTextureOrSingleColor(inputPins[2], constParams.factor);

    if (inTex1->isSingleColor() && inTex2->isSingleColor() && inTexFactor->isSingleColor())
    {
        Texture* outTex = nodeEvaluator->requestSingleColorTexture();
        outTex->setSingleColor(mixCols(inTex1->singleColor, inTex2->singleColor, inTexFactor->singleColor.r));

        outputPins[0].propagateTexture(outTex);
        return;
    }

    glm::ivec2 outRes = Texture::getFirstResolution({ inTex1, inTex2, inTexFactor });

    Texture* outTex = nodeEvaluator->requestTexture(outRes);

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(outRes, blockSize);
    kernMix<<<blocksPerGrid, blockSize>>>(*inTex1, *inTex2, *inTexFactor, outRes, *outTex);

    outputPins[0].propagateTexture(outTex);
}
