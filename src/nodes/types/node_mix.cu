#include "hip/hip_runtime.h"
#include "node_mix.hpp"

#include "cuda_includes.hpp"

NodeMix::NodeMix()
    : Node("mix")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "factor").setSingleChannel();
    addPin(PinType::INPUT, "image 1");
    addPin(PinType::INPUT, "image 2");
}

bool NodeMix::drawPinBeforeExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::INPUT && pinNumber == 0) // factor
    {
        return NodeUI::Checkbox(constParams.clamp, "clamp");
    }

    return false;
}

bool NodeMix::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT || pin->hasEdge())
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // factor
        ImGui::SameLine();
        return NodeUI::FloatEdit(constParams.factor, 0.01f, 0.f, 1.f);
    case 1: // in color 1
        ImGui::SameLine();
        return NodeUI::ColorEdit4(constParams.color1);
    case 2: // in color 2
        ImGui::SameLine();
        return NodeUI::ColorEdit4(constParams.color2);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

__host__ __device__ glm::vec4 mixCols(glm::vec4 col1, glm::vec4 col2, float factor, bool clamp)
{
    if (clamp)
    {
        factor = glm::clamp(factor, 0.f, 1.f);
    }
    return glm::mix(col1, col2, factor);
}

__global__ void kernMix(Texture inTex1, Texture inTex2, Texture inTexFactor, bool clamp, Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outTex.resolution.x || y >= outTex.resolution.y)
    {
        return;
    }

    glm::vec4 col1 = inTex1.getColorClamp<TextureType::MULTI>(x, y);
    glm::vec4 col2 = inTex2.getColorClamp<TextureType::MULTI>(x, y);
    float factor = inTexFactor.getColorClamp<TextureType::SINGLE>(x, y);

    outTex.setColor<TextureType::MULTI>(x, y, mixCols(col1, col2, factor, clamp));
}

// should work for differing resolutions but that hasn't been tested yet
void NodeMix::_evaluate()
{
    Texture* inTexFactor = getPinTextureOrUniformColor(inputPins[0], constParams.factor);
    Texture* inTex1 = getPinTextureOrUniformColor(inputPins[1], ColorUtils::srgbToLinear(constParams.color1));
    Texture* inTex2 = getPinTextureOrUniformColor(inputPins[2], ColorUtils::srgbToLinear(constParams.color2));

    if (inTex1->isUniform() && inTex2->isUniform() && inTexFactor->isUniform())
    {
        Texture* outTex = nodeEvaluator->requestUniformTexture();
        outTex->setUniformColor(mixCols(
            inTex1->getUniformColor<TextureType::MULTI>(),
            inTex2->getUniformColor<TextureType::MULTI>(),
            inTexFactor->getUniformColor<TextureType::SINGLE>(),
            constParams.clamp
        ));

        outputPins[0].propagateTexture(outTex);
        return;
    }

    glm::ivec2 outRes = Texture::getFirstResolutionFromList({ inTex1, inTex2, inTexFactor });
    Texture* outTex = nodeEvaluator->requestTexture<TextureType::MULTI>(outRes);

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_2D_X, DEFAULT_BLOCK_SIZE_2D_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(outRes, blockSize);
    kernMix<<<blocksPerGrid, blockSize>>>(*inTex1, *inTex2, *inTexFactor, constParams.clamp, *outTex);

    outputPins[0].propagateTexture(outTex);
}
