#include "node_fileinput.hpp"

#include "cuda_includes.hpp"

#include "stb_image.h"
#include "tinyexr.h"
#include <filesystem>

std::vector<const char*> NodeFileInput::colorSpaceOptions = { "linear", "sRGB" };

NodeFileInput::NodeFileInput()
    : Node("file input")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "color space").setNoConnect();
}

void NodeFileInput::reloadFile()
{
    if (texFile != nullptr) {
        --texFile->numReferences;
        texFile = nullptr;
    }

    bool isExr = isFileExr();

    float* host_pixels = nullptr;
    int width, height;
    if (isExr)
    {
        const char* err = nullptr;

        int ret = LoadEXR(&host_pixels, &width, &height, filePath.c_str(), &err);

        if (ret != TINYEXR_SUCCESS)
        {
            if (err)
            {
                fprintf(stderr, "ERR : %s\n", err);
                FreeEXRErrorMessage(err);
            }

            return;
        }
    }
    else
    {
        stbi_ldr_to_hdr_gamma(selectedColorSpace == 0 ? 1.0f : 2.2f); // 1.0f if linear, 2.2f if sRGB

        int channels;
        host_pixels = stbi_loadf(filePath.c_str(), &width, &height, &channels, 4);
    }

    if (host_pixels == nullptr) {
        return;
    }

    texFile = nodeEvaluator->requestTexture(glm::ivec2(width, height));
    CUDA_CHECK(hipMemcpy(texFile->dev_pixels, host_pixels, width * height * 4 * sizeof(float), hipMemcpyHostToDevice));

    if (isExr)
    {
        free(host_pixels);
    }
    else
    {
        stbi_image_free(host_pixels);
    }
}

bool NodeFileInput::isFileExr() const
{
    return std::filesystem::path(filePath).extension().string() == ".exr";
}

bool NodeFileInput::drawPinExtras(const Pin* pin, int pinNumber)
{
    ImGui::SameLine();

    bool didParameterChange;

    if (pin->pinType == PinType::INPUT)
    {
        switch (pinNumber)
        {
        case 0: // color space
            didParameterChange = NodeUI::Dropdown(selectedColorSpace, colorSpaceOptions);
            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }
    else
    {
        switch (pinNumber)
        {
        case 0: // file input
            didParameterChange = NodeUI::FilePicker(&filePath);

            if (didParameterChange)
            {
                selectedColorSpace = isFileExr() ? 0 : 1; // linear if EXR, sRGB otherwise
            }

            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }

    if (didParameterChange) {
        needsReloadFile = true;
    }
    return didParameterChange;
}

void NodeFileInput::evaluate()
{
    if (needsReloadFile) {
        needsReloadFile = false;
        reloadFile();
    }

    Texture* outTex;
    if (texFile == nullptr)
    {
        outTex = nodeEvaluator->requestSingleColorTexture();
        outTex->setSingleColor(glm::vec4(0, 0, 0, 1));
    }
    else
    {
        outTex = texFile;
    }

    ++outTex->numReferences; // cache this texture
    outputPins[0].propagateTexture(outTex);
}

std::string NodeFileInput::debugGetSrcFileName() const
{
    return __FILE__;
}
