#include "node_fileinput.hpp"

#include "cuda_includes.hpp"

#include "stb_image.h"
#include <filesystem>

std::vector<const char*> NodeFileInput::colorSpaceOptions = { "linear", "sRGB" };

NodeFileInput::NodeFileInput()
    : Node("file input")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "color space").setNoConnect();
}

void NodeFileInput::reloadFile()
{
    if (texFile != nullptr) {
        --texFile->numReferences;
        texFile = nullptr;
    }

    if (selectedColorSpace == 0) // linear
    {
        stbi_ldr_to_hdr_gamma(1.0f);
    }
    else // sRGB
    {
        stbi_ldr_to_hdr_gamma(2.2f);
    }

    int width, height, channels;
    float* host_pixels = stbi_loadf(filePath.c_str(), &width, &height, &channels, 4);

    if (host_pixels == nullptr) {
        return;
    }

    texFile = nodeEvaluator->requestTexture(glm::ivec2(width, height));
    CUDA_CHECK(hipMemcpy(texFile->dev_pixels, host_pixels, width * height * 4 * sizeof(float), hipMemcpyHostToDevice));

    stbi_image_free(host_pixels);
}

bool NodeFileInput::drawPinExtras(const Pin* pin, int pinNumber)
{
    ImGui::SameLine();

    bool didParameterChange;

    if (pin->pinType == PinType::INPUT)
    {
        switch (pinNumber)
        {
        case 0: // color space
            didParameterChange = NodeUI::Dropdown(selectedColorSpace, colorSpaceOptions);
            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }
    else
    {
        switch (pinNumber)
        {
        case 0: // file input
            didParameterChange = NodeUI::FilePicker(&filePath);

            if (didParameterChange)
            {
                std::string extension = std::filesystem::path(filePath).extension().string();
                if (extension == ".exr")
                {
                    selectedColorSpace = 0; // linear
                }
                else
                {
                    selectedColorSpace = 1; // sRGB
                }
            }

            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }

    if (didParameterChange) {
        needsReloadFile = true;
    }
    return didParameterChange;
}

void NodeFileInput::evaluate()
{
    if (needsReloadFile) {
        needsReloadFile = false;
        reloadFile();
    }

    Texture* outTex;
    if (texFile == nullptr)
    {
        outTex = nodeEvaluator->requestSingleColorTexture();
        outTex->setSingleColor(glm::vec4(0, 0, 0, 1));
    }
    else
    {
        outTex = texFile;
    }

    ++outTex->numReferences; // cache this texture
    outputPins[0].propagateTexture(outTex);
}

std::string NodeFileInput::debugGetSrcFileName() const
{
    return __FILE__;
}
