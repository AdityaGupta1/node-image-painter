#include "hip/hip_runtime.h"
#include "node_fileinput.hpp"

#include "cuda_includes.hpp"

#include "stb_image.h"
#include "tinyexr.h"
#include <filesystem>

std::vector<const char*> NodeFileInput::colorSpaceOptions = { "linear", "sRGB" };

NodeFileInput::NodeFileInput()
    : Node("file input")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "color space").setNoConnect();
}

unsigned int NodeFileInput::getTitleBarColor() const
{
    return IM_COL32(7, 94, 11, 255);
}

unsigned int NodeFileInput::getTitleBarSelectedColor() const
{
    return IM_COL32(47, 153, 53, 255);
}

__global__ void kernSrgbToLinear(Texture tex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= tex.resolution.x || y >= tex.resolution.y)
    {
        return;
    }

    int idx = y * tex.resolution.x + x;
    tex.setColor(idx, ColorUtils::srgbToLinear(tex.getColor(idx)));
}

void NodeFileInput::reloadFile()
{
    if (texFile != nullptr) {
        --texFile->numReferences;
        texFile = nullptr;
    }

    bool isExr = isFileExr();

    float* host_pixels = nullptr;
    int width, height;
    if (isExr)
    {
        const char* err = nullptr;

        int ret = LoadEXR(&host_pixels, &width, &height, filePath.c_str(), &err);

        if (ret != TINYEXR_SUCCESS)
        {
            if (err)
            {
                fprintf(stderr, "ERR : %s\n", err);
                FreeEXRErrorMessage(err);
            }

            return;
        }
    }
    else
    {
        stbi_ldr_to_hdr_gamma(selectedColorSpace == 0 ? 1.0f : 2.2f); // 1.0f if linear, 2.2f if sRGB

        int channels;
        host_pixels = stbi_loadf(filePath.c_str(), &width, &height, &channels, 4);
    }

    if (host_pixels == nullptr) {
        return;
    }

    texFile = nodeEvaluator->requestTexture(glm::ivec2(width, height));
    CUDA_CHECK(hipMemcpy(texFile->getDevPixels(), host_pixels, width * height * 4 * sizeof(float), hipMemcpyHostToDevice));

    if (isExr)
    {
        free(host_pixels);

        if (selectedColorSpace == 1) // sRGB
        {
            const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
            const dim3 blocksPerGrid = calculateNumBlocksPerGrid(texFile->resolution, blockSize);
            kernSrgbToLinear<<<blocksPerGrid, blockSize>>>(*texFile);
        }
    }
    else
    {
        stbi_image_free(host_pixels);
    }
}

bool NodeFileInput::isFileExr() const
{
    return std::filesystem::path(filePath).extension().string() == ".exr";
}

bool NodeFileInput::drawPinExtras(const Pin* pin, int pinNumber)
{
    ImGui::SameLine();

    bool didParameterChange;

    if (pin->pinType == PinType::INPUT)
    {
        switch (pinNumber)
        {
        case 0: // color space
            didParameterChange = NodeUI::Dropdown(selectedColorSpace, colorSpaceOptions);
            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }
    else
    {
        switch (pinNumber)
        {
        case 0: // file input
            didParameterChange = NodeUI::FilePicker(&filePath, { "Image Files (.png, .jpg, .jpeg, .exr)", "*.png *.jpg *.jpeg *.exr" });

            if (didParameterChange)
            {
                selectedColorSpace = isFileExr() ? 0 : 1; // linear if EXR, sRGB otherwise
            }

            break;
        default:
            throw std::runtime_error("invalid pin number");
        }
    }

    if (didParameterChange) {
        needsReloadFile = true;
    }
    return didParameterChange;
}

void NodeFileInput::_evaluate()
{
    if (needsReloadFile) {
        reloadFile();
    }

    Texture* outTex;
    if (texFile == nullptr)
    {
        outTex = nodeEvaluator->requestSingleColorTexture();
        outTex->setSingleColor(glm::vec4(0, 0, 0, 1));
    }
    else
    {
        outTex = texFile;
    }

    if (needsReloadFile)
    {
        ++outTex->numReferences; // cache this texture; numReferences is decremented by reloadFile()
        needsReloadFile = false;
    }
    outputPins[0].propagateTexture(outTex);
}
