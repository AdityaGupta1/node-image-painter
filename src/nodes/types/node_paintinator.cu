#include "hip/hip_runtime.h"
#include "node_paintinator.hpp"

#include "cuda_includes.hpp"

#include "random_utils.hpp"
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <glm/gtx/component_wise.hpp>
#include <glm/gtc/constants.hpp>

#include "stb_image.h"

#include "npp_includes.hpp"

bool NodePaintinator::hasLoadedBrushes = false;
hipArray_t NodePaintinator::brushPixelArray;
hipTextureObject_t NodePaintinator::brushTextureObj;

NodePaintinator::NodePaintinator()
    : Node("paint-inator")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "image");
    addPin(PinType::INPUT, "num strokes").setNoConnect();
    addPin(PinType::INPUT, "min stroke size").setNoConnect();
    addPin(PinType::INPUT, "max stroke size").setNoConnect();
    addPin(PinType::INPUT, "size bias").setNoConnect();

    setExpensive();
}

void NodePaintinator::freeDeviceMemory()
{
    hipDestroyTextureObject(brushTextureObj);
    hipFreeArray(brushPixelArray);
}

bool NodePaintinator::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT || pin->hasEdge())
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // image
        return false;
    case 1: // num strokes
        ImGui::SameLine();
        return NodeUI::IntEdit(backupNumStrokes, 0.02f, 9, 19);
    case 2: // min stroke size
        ImGui::SameLine();
        return NodeUI::IntEdit(backupMinStrokeSize, 0.15f, 10, backupMaxStrokeSize);
    case 3: // max stroke size
        ImGui::SameLine();
        return NodeUI::IntEdit(backupMaxStrokeSize, 0.15f, backupMinStrokeSize, 1000);
    case 4: // size bias
        ImGui::SameLine();
        return NodeUI::FloatEdit(backupSizeBias, 0.01f, 0.01f, 100.f);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

struct PaintStroke
{
    glm::ivec2 pos;
    glm::mat2 matRotate;
    float scale;
    glm::vec3 color;
    glm::vec2 uv;
};

struct PaintStrokeComparator
{
    __host__ __device__ bool operator()(const PaintStroke& stroke1, const PaintStroke& stroke2)
    {
        return stroke1.scale < stroke2.scale;
    }
};

/*
__global__ void kernGeneratePaintStrokes(Texture inTex, PaintStroke* strokes, int numStrokes, int minStrokeSize, int maxStrokeSize, float sizeBias)
{
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx >= numStrokes)
    {
        return;
    }

    thrust::default_random_engine rng = makeSeededRandomEngine(idx);
    thrust::uniform_int_distribution<int> distX(0, inTex.resolution.x - 1);
    thrust::uniform_int_distribution<int> distY(0, inTex.resolution.y - 1);
    glm::ivec2 pos(distX(rng), distY(rng));

    thrust::uniform_real_distribution<float> u01(0, 1);
    float sinVal, cosVal;
    sincosf(u01(rng) * glm::two_pi<float>(), &sinVal, &cosVal);
    glm::mat2 matRotate(cosVal, sinVal, -sinVal, cosVal);

    float scale = minStrokeSize + (maxStrokeSize - minStrokeSize) * powf(u01(rng), sizeBias);

    glm::vec3 color(inTex.dev_pixels[pos.y * inTex.resolution.x + pos.x]);

    thrust::uniform_int_distribution<int> distUv(0, 3);
    glm::vec2 uv(distUv(rng) * 0.25f, distUv(rng) * 0.25f);

    strokes[idx] = { glm::vec2(pos) + glm::vec2(0.5f), matRotate, scale, color, uv };
}
*/

void NodePaintinator::loadBrushes()
{
    int width, height, channels;
    unsigned char* host_pixels = stbi_load("assets/brushes/test.png", &width, &height, &channels, 4);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    int pitch = width * sizeof(uchar4);

    CUDA_CHECK(hipMallocArray(
        &brushPixelArray,
        &channelDesc,
        width,
        height
    ));

    CUDA_CHECK(hipMemcpy2DToArray(brushPixelArray,
        0, // wOffset
        0, // hOffset
        host_pixels,
        pitch,
        pitch,
        height,
        hipMemcpyHostToDevice
    ));

    stbi_image_free(host_pixels);

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = brushPixelArray;

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;
    texDesc.maxAnisotropy = 1;
    texDesc.maxMipmapLevelClamp = 99;
    texDesc.minMipmapLevelClamp = 0;
    texDesc.mipmapFilterMode = hipFilterModePoint;
    texDesc.borderColor[0] = 1.0f;
    texDesc.sRGB = 0;

    CUDA_CHECK(hipCreateTextureObject(&brushTextureObj, &resDesc, &texDesc, nullptr));

    hasLoadedBrushes = true;
}

__global__ void kernFillEmptyTexture(Texture tex, int numPixels)
{
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx >= numPixels)
    {
        return;
    }

    tex.dev_pixels[idx] = glm::vec4(0, 0, 0, 0);
}

__global__ void kernCalculateColorDifference(Texture paintedTex, Texture refTex, float* colorDiff, int numPixels)
{
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx >= numPixels)
    {
        return;
    }

    glm::vec4 paintedCol = paintedTex.dev_pixels[idx];

    float diff;
    if (paintedCol.a == 0.f)
    {
        diff = 1e20f; // big number but not FLT_MAX to avoid overflow issues when summing error
    }
    else
    {
        diff = glm::distance(glm::vec3(paintedCol), glm::vec3(refTex.dev_pixels[idx]));
    }

    colorDiff[idx] = diff;
}

// I doubt this has coalesced memory accesses, which is probably not a good thing
__global__ void kernPrepareStrokes(Texture refTex, PaintStroke* strokes, int numStrokes)
{
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx >= numStrokes)
    {
        return;
    }

    PaintStroke& stroke = strokes[idx];

    thrust::default_random_engine rng = thrust::default_random_engine(hash(idx) ^ hash(numStrokes));
    thrust::uniform_real_distribution<float> u01(0, 1);
    float sinVal, cosVal;
    sincosf(u01(rng) * glm::two_pi<float>(), &sinVal, &cosVal);
    stroke.matRotate = { cosVal, sinVal, -sinVal, cosVal };

    stroke.color = glm::vec3(refTex.dev_pixels[stroke.pos.y * refTex.resolution.x + stroke.pos.x]);

    thrust::uniform_int_distribution<int> distUv(0, 3);
    stroke.uv = glm::vec2(distUv(rng) * 0.25f, distUv(rng) * 0.25f);
}

#define NUM_SHARED_STROKES 512

__global__ void kernPaint(Texture outTex, PaintStroke* strokes, int numStrokes, hipTextureObject_t brushTex)
{
    __shared__ PaintStroke shared_strokes[NUM_SHARED_STROKES];
    __shared__ int shared_numFinishedThreads;

    const int localIdx = threadIdx.y * blockDim.x + threadIdx.x;

    if (localIdx == 0)
    {
        shared_numFinishedThreads = 0;
    }

    __syncthreads();

    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    const bool inBounds = x < outTex.resolution.x&& y < outTex.resolution.y;

    if (!inBounds)
    {
        atomicAdd(&shared_numFinishedThreads, 1);
    }

    bool hasColor = false;
    glm::vec4 topColor = glm::vec4(0, 0, 0, 0);
    glm::vec2 thisPos = glm::vec2(x, y);

    int strokesStart = 0;
    const int numTotalThreads = blockDim.x * blockDim.y;
    while (shared_numFinishedThreads != numTotalThreads && strokesStart < numStrokes)
    {
        if (localIdx < NUM_SHARED_STROKES)
        {
            // no issues with indices going out of bounds if numStrokes is a multiple of NUM_SHARED_STROKES
            shared_strokes[localIdx] = strokes[strokesStart + localIdx];
        }

        strokesStart += NUM_SHARED_STROKES;

        __syncthreads();

        if (inBounds && !hasColor)
        {
            for (int strokeIdx = 0; strokeIdx < NUM_SHARED_STROKES; ++strokeIdx)
            {
                const PaintStroke& stroke = shared_strokes[strokeIdx];

                glm::vec2 localPos = stroke.matRotate * (thisPos - glm::vec2(stroke.pos));
                if (glm::compMax(glm::abs(localPos)) > stroke.scale)
                {
                    continue;
                }

                glm::vec2 uv = ((localPos / stroke.scale) + 1.f) * 0.5f;

                uv = stroke.uv + uv * 0.25f;
                float4 bottomColor = tex2D<float4>(brushTex, uv.x, uv.y);
                if (bottomColor.w == 0.f)
                {
                    continue;
                }

                // probably not how real paint mixes but whatever
                glm::vec3 bottomRgb = glm::vec3(bottomColor.x, bottomColor.y, bottomColor.z) * stroke.color;
                float newAlpha = bottomColor.w + ((1.f - bottomColor.w) * topColor.a);
                topColor = glm::vec4(glm::mix(bottomRgb, glm::vec3(topColor), topColor.a), newAlpha);

                if (topColor.a > 0.999f)
                {
                    topColor.a = 1.f;
                    hasColor = true;
                    atomicAdd(&shared_numFinishedThreads, 1);
                    break;
                }
            }
        }

        __syncthreads();
    }

    if (!inBounds)
    {
        return;
    }

    const int idx = y * outTex.resolution.x + x;
    // TODO: read existing color and blend accordingly
    if (topColor.a != 0.f)
    {
        outTex.dev_pixels[idx] = topColor;
    }
}

// TODO: make these into node parameters
static const int numLayers = 5;
static const float gridSizeFactor = 0.3f;
static const float newStrokeErrorThreshold = 0.3f;

// reference paper: https://dl.acm.org/doi/10.1145/280814.280951
void NodePaintinator::evaluate()
{
    Texture* inTex = getPinTextureOrSingleColor(inputPins[0], glm::vec4(0, 0, 0, 1));

    if (inTex->isSingleColor())
    {
        outputPins[0].propagateTexture(inTex);
        return;
    }

    if (!hasLoadedBrushes)
    {
        loadBrushes();
    }

    Texture* outTex = nodeEvaluator->requestTexture(inTex->resolution);

    const int numPixels = outTex->resolution.x * outTex->resolution.y;
    const dim3 blockSize1d(256);
    const dim3 blocksPerGrid1d(calculateNumBlocksPerGrid(numPixels, blockSize1d.x));

    kernFillEmptyTexture<<<blocksPerGrid1d, blockSize1d>>>(
        *outTex, numPixels
    );

    /*
    PaintStroke* dev_strokes;
    const int numStrokes = 1 << backupNumStrokes;
    CUDA_CHECK(hipMalloc(&dev_strokes, numStrokes * sizeof(PaintStroke))); // TODO: malloc once and re-malloc only if numStrokes changes

    const dim3 blockSize1d(256);
    const dim3 blocksPerGrid1d(calculateNumBlocksPerGrid(numStrokes, blockSize1d.x));

    kernGeneratePaintStrokes<<<blocksPerGrid1d, blockSize1d>>>(
        *inTex,
        dev_strokes,
        numStrokes,
        backupMinStrokeSize,
        backupMaxStrokeSize,
        1.f / backupSizeBias
    );

    thrust::sort(thrust::device, dev_strokes, dev_strokes + numStrokes, PaintStrokeComparator());

    const dim3 blockSize2d(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid2d = calculateNumBlocksPerGrid(inTex->resolution, blockSize2d);

    kernPaint<<<blocksPerGrid2d, blockSize2d>>>(
        *outTex,
        dev_strokes,
        numStrokes,
        brushTextureObj
    );

    CUDA_CHECK(hipFree(dev_strokes));

    outputPins[0].propagateTexture(outTex);
    */

    Texture* scratchTex = nodeEvaluator->requestTexture(inTex->resolution);
    Texture* refTex = nodeEvaluator->requestTexture(inTex->resolution);

    const int width = inTex->resolution.x;
    const int height = inTex->resolution.y;
    NppiSize oSrcSize = { width, height };
    NppiPoint oSrcOffset = { 0, 0 };

    NppiSize oSizeROI = { width, height };

    float* host_colorDiff = new float[numPixels];
    float* dev_colorDiff;
    CUDA_CHECK(hipMalloc(&dev_colorDiff, numPixels * sizeof(float)));

    const dim3 blockSize2d(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid2d = calculateNumBlocksPerGrid(inTex->resolution, blockSize2d);

    float logMinStrokeSize = logf(backupMinStrokeSize);
    float logMaxStrokeSize = logf(backupMaxStrokeSize);
    for (int i = 0; i < numLayers; ++i)
    {
        // =========================
        // MAKE REFERENCE IMAGE
        // =========================

        float logStrokeSize = glm::mix(logMaxStrokeSize, logMinStrokeSize, (float)i / std::max(numLayers - 1, 1));
        float strokeSize = expf(logStrokeSize);

        const int kernelRadius = (int)strokeSize; // TODO: check that this is correct and not off by a factor of 2
        const int kernelDiameter = kernelRadius * 2 + 1;

        // TODO: malloc space for all kernels at once and fill them all using one kernel invocation
        //       should significantly reduce the number of calls to hipMalloc
        float* host_kernel = new float[kernelDiameter];
        float* dev_kernel;
        CUDA_CHECK(hipMalloc(&dev_kernel, kernelDiameter * sizeof(float)));

        const float sigma = kernelDiameter / 9.f;
        const float sigma2 = sigma * sigma;
        const float normalizationFactor = 1.f / sqrtf(2 * glm::pi<float>() * sigma2);
        const float exponentFactor = -1.f / (2.f * sigma2);

        for (int i = 0; i < kernelDiameter; ++i)
        {
            int x = i - kernelRadius;
            host_kernel[i] = normalizationFactor * expf(exponentFactor * x * x);
        }
        hipMemcpy(dev_kernel, host_kernel, kernelDiameter * sizeof(float), hipMemcpyHostToDevice);

        Npp32s nMaskSize = kernelDiameter;
        Npp32s nAnchor = kernelRadius;

        NPP_CHECK(
            nppiFilterColumnBorder_32f_C4R(
                (Npp32f*)inTex->dev_pixels, width * 4 * sizeof(float),
                oSrcSize, oSrcOffset,
                (Npp32f*)scratchTex->dev_pixels, width * 4 * sizeof(float),
                oSizeROI,
                (Npp32f*)dev_kernel, nMaskSize, nAnchor,
                NPP_BORDER_REPLICATE)
        );

        NPP_CHECK(
            nppiFilterRowBorder_32f_C4R(
                (Npp32f*)scratchTex->dev_pixels, width * 4 * sizeof(float),
                oSrcSize, oSrcOffset,
                (Npp32f*)refTex->dev_pixels, width * 4 * sizeof(float),
                oSizeROI,
                (Npp32f*)dev_kernel, nMaskSize, nAnchor,
                NPP_BORDER_REPLICATE)
        );

        delete[] host_kernel;
        CUDA_CHECK(hipFree(dev_kernel));

        // =========================
        // PAINT LAYER
        // =========================

        kernCalculateColorDifference<<<blocksPerGrid1d, blockSize1d>>>(
            *outTex, *refTex, dev_colorDiff, numPixels
        );

        CUDA_CHECK(hipMemcpy(host_colorDiff, dev_colorDiff, numPixels * sizeof(float), hipMemcpyDeviceToHost));

        // gridSize is always even and at least 2
        int gridSize = (int)(strokeSize * 2 * gridSizeFactor);
        if (gridSize % 2 != 0)
        {
            --gridSize;
        }
        gridSize = std::max(gridSize, 2);
        int halfGridSize = gridSize / 2;

        std::vector<PaintStroke> host_strokes;
        for (int cellY = 0; cellY < height + halfGridSize; cellY += gridSize)
        {
            for (int cellX = 0; cellX < width + halfGridSize; cellX += gridSize)
            {
                int xMin = std::max(cellX - halfGridSize, 0);
                int xMax = std::min(cellX + halfGridSize, width);
                int yMin = std::max(cellY - halfGridSize, 0);
                int yMax = std::min(cellY + halfGridSize, height);

                int gridPixels = (xMax - xMin) * (yMax - yMin);

                float totalError = 0.f;
                float maxError = -FLT_MAX;
                glm::ivec2 maxErrorPos;
                for (int y = yMin; y < yMax; ++y)
                {
                    for (int x = xMin; x < xMax; ++x)
                    {
                        float error = host_colorDiff[y * width + x];
                        totalError += error;
                        if (error > maxError)
                        {
                            maxError = error;
                            maxErrorPos = glm::ivec2(x, y);
                        }
                    }
                }

                float areaError = totalError / gridPixels;
                if (areaError < newStrokeErrorThreshold)
                {
                    continue;
                }

                PaintStroke newStroke;
                newStroke.pos = maxErrorPos;
                newStroke.scale = strokeSize;
                // other fields are set by kernPrepareStrokes
                host_strokes.push_back(newStroke);
            }
        }

        // TODO: hipMalloc dev_strokes only once based on maximum number of strokes for a layer
        PaintStroke* dev_strokes;
        const int numStrokes = host_strokes.size();
        CUDA_CHECK(hipMalloc(&dev_strokes, numStrokes * sizeof(PaintStroke)));
        CUDA_CHECK(hipMemcpy(dev_strokes, host_strokes.data(), numStrokes * sizeof(PaintStroke), hipMemcpyHostToDevice));

        kernPrepareStrokes<<<blocksPerGrid1d, blockSize1d>>>(
            *refTex, dev_strokes, numStrokes
        );

        kernPaint<<<blocksPerGrid2d, blockSize2d>>>(
            *outTex, dev_strokes, numStrokes, brushTextureObj
        );

        CUDA_CHECK(hipFree(dev_strokes));
    }

    delete[] host_colorDiff;
    CUDA_CHECK(hipFree(dev_colorDiff));

    outputPins[0].propagateTexture(outTex);
}

std::string NodePaintinator::debugGetSrcFileName() const
{
    return __FILE__;
}

