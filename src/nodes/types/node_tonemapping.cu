#include "hip/hip_runtime.h"
#include "node_tonemapping.hpp"

#include "cuda_includes.hpp"

std::vector<const char*> NodeToneMapping::toneMappingOptions = { "none", "AgX", "AgX (golden)", "AgX (punchy)", "reinhard", "ACES filmic" };

NodeToneMapping::NodeToneMapping()
    : Node("tone mapping")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "image");
    addPin(PinType::INPUT, "tone mapping").setNoConnect();
}

unsigned int NodeToneMapping::getTitleBarColor() const
{
    return IM_COL32(130, 0, 0, 255);
}

unsigned int NodeToneMapping::getTitleBarSelectedColor() const
{
    return IM_COL32(190, 0, 0, 255);
}

bool NodeToneMapping::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT)
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // image
        return false;
    case 1: // tone mapping
        ImGui::SameLine();
        return NodeUI::Dropdown(selectedToneMapping, toneMappingOptions);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

__host__ __device__ glm::vec4 applyToneMapping(glm::vec4 col, int toneMapping)
{
    glm::vec3 rgb = glm::max(glm::vec3(col), 0.f);

    switch (toneMapping)
    {
    case 0:
        break;
    case 1:
        rgb = ColorUtils::AgX(rgb, 0);
        break;
    case 2:
        rgb = ColorUtils::AgX(rgb, 1);
        break;
    case 3:
        rgb = ColorUtils::AgX(rgb, 2);
        break;
    case 4:
        rgb = ColorUtils::reinhard(rgb);
        break;
    case 5:
        rgb = ColorUtils::ACESFilm(rgb);
        break;
    }

    return glm::vec4(rgb, col.a);
}

__global__ void kernApplyToneMapping(Texture inTex, int toneMapping, Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= outTex.resolution.x || y >= outTex.resolution.y)
    {
        return;
    }

    const int idx = y * inTex.resolution.x + x;
    outTex.dev_pixels[idx] = applyToneMapping(inTex.dev_pixels[idx], toneMapping);
}

void NodeToneMapping::evaluate()
{
    Texture* inTex = inputPins[0].getSingleTexture();

    if (inTex->isSingleColor())
    {
        Texture* outTex = nodeEvaluator->requestSingleColorTexture();
        outTex->setSingleColor(applyToneMapping(inTex->singleColor, selectedToneMapping));
        outputPins[0].propagateTexture(outTex);
        return;
    }

    Texture* outTex = nodeEvaluator->requestTexture(inTex->resolution);

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(inTex->resolution, blockSize);
    kernApplyToneMapping<<<blocksPerGrid, blockSize>>>(*inTex, selectedToneMapping, *outTex);

    outputPins[0].propagateTexture(outTex);
}
