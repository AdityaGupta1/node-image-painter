#include "hip/hip_runtime.h"
#include "node_exposure.hpp"

#include "cuda_includes.hpp"

NodeExposure::NodeExposure()
    : Node("exposure")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "image");
    addPin(PinType::INPUT, "exposure").setNoConnection();
}

__global__ void kernExposure(Texture inTex, float multiplier, Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= inTex.resolution.x || y >= inTex.resolution.y)
    {
        return;
    }

    int idx = y * inTex.resolution.x + x;
    glm::vec4 col = inTex.dev_pixels[idx];
    outTex.dev_pixels[idx] = glm::vec4(glm::vec3(col) * multiplier, col.a);
}

bool NodeExposure::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT || pin->hasEdge())
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // image
        ImGui::SameLine();
        return NodeUI::ColorEdit4(backupCol);
    case 1: // exposure
        ImGui::SameLine();
        return NodeUI::FloatEdit(backupExposure, 0.01f);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

void NodeExposure::evaluate()
{
    Texture* inTex = getPinTextureOrSingleColor(inputPins[0], ColorUtils::srgbToLinear(backupCol));

    if (inTex->isSingleColor()) {
        Texture* outTex = nodeEvaluator->requestSingleColorTexture();

        if (backupExposure == 0.f) {
            outTex->setSingleColor(inTex->singleColor);
        }
        else
        {
            glm::vec4 outCol = glm::vec4(glm::vec3(inTex->singleColor) * powf(2.f, backupExposure), inTex->singleColor.a);
            outTex->setSingleColor(outCol);
        }

        outputPins[0].propagateTexture(outTex);
        return;
    }

    // inTex is not a single color
    if (backupExposure == 0.0f) {
        outputPins[0].propagateTexture(inTex);
        return;
    }

    // inTex is not a single color and backupExposure != 0.f
    Texture* outTex = nodeEvaluator->requestTexture(inTex->resolution);

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid = calculateBlocksPerGrid(inTex->resolution, blockSize);
    kernExposure<<<blocksPerGrid, blockSize>>>(*inTex, powf(2.f, backupExposure), *outTex);

    outputPins[0].propagateTexture(outTex);
}

std::string NodeExposure::debugGetSrcFileName() const
{
    return __FILE__;
}

