#include "hip/hip_runtime.h"
#include "node_exposure.hpp"

#include "cuda_includes.hpp"

NodeExposure::NodeExposure()
    : Node("exposure")
{
    addPin(PinType::OUTPUT, "image");

    addPin(PinType::INPUT, "image");
    addPin(PinType::INPUT, "exposure").setNoConnect();
}

__global__ void kernExposure(Texture inTex, float multiplier, Texture outTex)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= inTex.resolution.x || y >= inTex.resolution.y)
    {
        return;
    }

    int idx = y * inTex.resolution.x + x;
    glm::vec4 col = inTex.getColor<TextureType::MULTI>(idx);
    outTex.setColor<TextureType::MULTI>(idx, glm::vec4(glm::vec3(col) * multiplier, col.a));
}

bool NodeExposure::drawPinExtras(const Pin* pin, int pinNumber)
{
    if (pin->pinType == PinType::OUTPUT || pin->hasEdge())
    {
        return false;
    }

    switch (pinNumber)
    {
    case 0: // image
        ImGui::SameLine();
        return NodeUI::ColorEdit4(constParams.color);
    case 1: // exposure
        ImGui::SameLine();
        return NodeUI::FloatEdit(constParams.exposure, 0.01f);
    default:
        throw std::runtime_error("invalid pin number");
    }
}

void NodeExposure::_evaluate()
{
    Texture* inTex = getPinTextureOrUniformColor(inputPins[0], ColorUtils::srgbToLinear(constParams.color));

    if (inTex->isUniform()) {
        Texture* outTex = nodeEvaluator->requestUniformTexture();

        if (constParams.exposure == 0.f) {
            outTex->setUniformColor(inTex->getUniformColor());
        }
        else
        {
            glm::vec4 outCol = glm::vec4(glm::vec3(inTex->getUniformColor()) * powf(2.f, constParams.exposure), inTex->getUniformColor().a);
            outTex->setUniformColor(outCol);
        }

        outputPins[0].propagateTexture(outTex);
        return;
    }

    // inTex is not uniform
    if (constParams.exposure == 0.f) {
        outputPins[0].propagateTexture(inTex);
        return;
    }

    // inTex is not uniform and constParams.exposure != 0.f
    Texture* outTex = nodeEvaluator->requestTexture<TextureType::MULTI>(inTex->resolution);

    const dim3 blockSize(DEFAULT_BLOCK_SIZE_X, DEFAULT_BLOCK_SIZE_Y);
    const dim3 blocksPerGrid = calculateNumBlocksPerGrid(inTex->resolution, blockSize);
    kernExposure<<<blocksPerGrid, blockSize>>>(*inTex, powf(2.f, constParams.exposure), *outTex);

    outputPins[0].propagateTexture(outTex);
}
